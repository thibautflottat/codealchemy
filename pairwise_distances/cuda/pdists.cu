#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void pairwise_distances_kernel(int n_particles, float* positions, float* distances) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n_particles && j < n_particles && i < j) {
        float L = 1.0f;
        float half_L = 0.5f;
        float dx = positions[i * 3] - positions[j * 3];
        float dy = positions[i * 3 + 1] - positions[j * 3 + 1];
        float dz = positions[i * 3 + 2] - positions[j * 3 + 2];

        dx = (dx > half_L) ? dx - L : (dx < -half_L) ? dx + L : dx;
        dy = (dy > half_L) ? dy - L : (dy < -half_L) ? dy + L : dy;
        dz = (dz > half_L) ? dz - L : (dz < -half_L) ? dz + L : dz;

        float dist = sqrtf(dx * dx + dy * dy + dz * dz);
        distances[i * n_particles + j] = dist;
        distances[j * n_particles + i] = dist;
    }
}

void pairwise_distances(int n_particles, float* d_positions, float* d_distances) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n_particles + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (n_particles + threadsPerBlock.y - 1) / threadsPerBlock.y);

    pairwise_distances_kernel<<<numBlocks, threadsPerBlock>>>(n_particles, d_positions, d_distances);
    hipDeviceSynchronize();
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <number_of_particles>" << std::endl;
        return 1;
    }

    int n_particles = std::atoi(argv[1]);
    if (n_particles <= 0) {
        std::cerr << "Number of particles must be a positive integer." << std::endl;
        return 1;
    }

    size_t positions_size = n_particles * 3 * sizeof(float);
    size_t distances_size = n_particles * n_particles * sizeof(float);

    float* h_positions = (float*)malloc(positions_size);
    float* h_distances = (float*)malloc(distances_size);

    // Initialize positions with random values
    for (int i = 0; i < n_particles * 3; i++) {
        h_positions[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    float* d_positions;
    float* d_distances;

    hipMalloc(&d_positions, positions_size);
    hipMalloc(&d_distances, distances_size);

    hipMemcpy(d_positions, h_positions, positions_size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    pairwise_distances(n_particles, d_positions, d_distances);
    hipMemcpy(h_distances, d_distances, distances_size, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    std::cout << "Execution time (milliseconds): " << milliseconds << std::endl;

    // Debugging: Print part of the distances matrix
    // for (int i = 0; i < std::min(n_particles, 10); i++) {
    //     for (int j = 0; j < std::min(n_particles, 10); j++) {
    //         std::cout << "distances(" << i << ", " << j << ") = " << h_distances[i * n_particles + j] << std::endl;
    //     }
    // }

    free(h_positions);
    free(h_distances);
    hipFree(d_positions);
    hipFree(d_distances);

    return 0;
}
