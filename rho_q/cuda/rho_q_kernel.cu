#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <cmath>

__global__ void rho_q_kernel(const double* x, const double* q, thrust::complex<double>* rho, size_t Nx, size_t Nq) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < Nq) {
        thrust::complex<double> rho_value(0.0, 0.0);

        for (size_t j = 0; j < Nx; ++j) {
            double alpha = x[j * 3] * q[i * 3] + x[j * 3 + 1] * q[i * 3 + 1] + x[j * 3 + 2] * q[i * 3 + 2];
            rho_value += thrust::exp(thrust::complex<double>(0.0, alpha));
        }

        rho[i] = rho_value;
    }
}

extern "C" void rho_q_cuda(const double* x, const double* q, thrust::complex<double>* rho, size_t Nx, size_t Nq) {
    double *d_x, *d_q;
    thrust::complex<double>* d_rho;

    // Allocate device memory
    hipMalloc(&d_x, Nx * 3 * sizeof(double));
    hipMalloc(&d_q, Nq * 3 * sizeof(double));
    hipMalloc(&d_rho, Nq * sizeof(thrust::complex<double>));

    // Copy data to device
    hipMemcpy(d_x, x, Nx * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_q, q, Nq * 3 * sizeof(double), hipMemcpyHostToDevice);

    // Set up the execution configuration
    int blockSize = 256;
    int numBlocks = (Nq + blockSize - 1) / blockSize;

    // Launch the kernel
    rho_q_kernel<<<numBlocks, blockSize>>>(d_x, d_q, d_rho, Nx, Nq);

    // Copy the result back to the host
    hipMemcpy(rho, d_rho, Nq * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_q);
    hipFree(d_rho);
}
