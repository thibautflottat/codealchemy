#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <random>
#include <chrono>

// CUDA kernel to compute rho_q on the GPU
__global__ void rho_q_kernel(const double* x, const double* q, thrust::complex<double>* rho, size_t Nx, size_t Nq) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < Nq) {
        thrust::complex<double> rho_value(0.0, 0.0);

        for (size_t j = 0; j < Nx; ++j) {
            double alpha = x[j * 3] * q[i * 3] + x[j * 3 + 1] * q[i * 3 + 1] + x[j * 3 + 2] * q[i * 3 + 2];
            rho_value += thrust::exp(thrust::complex<double>(0.0, alpha));
        }

        rho[i] = rho_value;
    }
}

// Function to set up and launch the CUDA kernel
void rho_q_cuda(const std::vector<double>& x, const std::vector<double>& q, std::vector<std::complex<double>>& rho, size_t Nx, size_t Nq) {
    double *d_x, *d_q;
    thrust::complex<double>* d_rho;

    // Allocate device memory
    hipMalloc(&d_x, Nx * 3 * sizeof(double));
    hipMalloc(&d_q, Nq * 3 * sizeof(double));
    hipMalloc(&d_rho, Nq * sizeof(thrust::complex<double>));

    // Copy data to device
    hipMemcpy(d_x, x.data(), Nx * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_q, q.data(), Nq * 3 * sizeof(double), hipMemcpyHostToDevice);

    // Set up the execution configuration
    int blockSize = 256;
    int numBlocks = (Nq + blockSize - 1) / blockSize;

    // Launch the kernel
    rho_q_kernel<<<numBlocks, blockSize>>>(d_x, d_q, d_rho, Nx, Nq);

    // Synchronize to ensure all threads have finished
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(rho.data(), d_rho, Nq * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_q);
    hipFree(d_rho);
}

int main() {
    size_t Nx = 100000;
    size_t Nq = 100000;

    std::vector<double> x(Nx * 3);
    std::vector<double> q(Nq * 3);
    std::vector<std::complex<double>> rho(Nq, 0.0);

    // Generate random data for x and q
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);

    for (size_t i = 0; i < Nx * 3; ++i) {
        x[i] = dis(gen);
    }

    for (size_t i = 0; i < Nq * 3; ++i) {
        q[i] = dis(gen);
    }

    // Time the CUDA execution
    auto start_time = std::chrono::high_resolution_clock::now();
    rho_q_cuda(x, q, rho, Nx, Nq);
    auto end_time = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << "CUDA Execution Time: " << elapsed.count() << " seconds" << std::endl;

    // Output a sample result
    std::cout << "rho[0]: " << rho[0] << std::endl;

    return 0;
}
