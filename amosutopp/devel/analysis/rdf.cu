#include "hip/hip_runtime.h"
#include "rdf.hpp"
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

// GPU kernel to compute RDF
__global__ void compute_rdf_kernel(const double* positions, double* rdf, double bin_size, int num_bins, int n_particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_particles) return;

    for (int j = i + 1; j < n_particles; ++j) {
        double dx = positions[3*i] - positions[3*j];
        double dy = positions[3*i + 1] - positions[3*j + 1];
        double dz = positions[3*i + 2] - positions[3*j + 2];

        double r = sqrt(dx*dx + dy*dy + dz*dz);
        int bin = static_cast<int>(r / bin_size);
        if (bin < num_bins) {
            atomicAdd(&rdf[bin], 2.0);
        }
    }
}

void compute_rdf_gpu(const std::vector<double>& positions, std::vector<double>& rdf, double bin_size, int num_bins) {
    size_t n_particles = positions.size() / 3;
    double* d_positions;
    double* d_rdf;

    // Allocate GPU memory
    hipMalloc(&d_positions, positions.size() * sizeof(double));
    hipMalloc(&d_rdf, num_bins * sizeof(double));

    // Copy data to GPU
    hipMemcpy(d_positions, positions.data(), positions.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_rdf, 0, num_bins * sizeof(double));

    // Launch the RDF kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n_particles + threadsPerBlock - 1) / threadsPerBlock;
    compute_rdf_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_positions, d_rdf, bin_size, num_bins, n_particles);

    // Copy results back to host
    hipMemcpy(rdf.data(), d_rdf, num_bins * sizeof(double), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_positions);
    hipFree(d_rdf);
}